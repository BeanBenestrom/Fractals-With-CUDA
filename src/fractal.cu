#include "hip/hip_runtime.h"
// #include <complex>
#include "fractal.cuh"
#include <iostream>


__global__ void kernel(int w, int h, PERSISION x, PERSISION y, PERSISION zoom, PERSISION var_x, PERSISION var_y, bool mandel, int* d_outPut)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < w*h) 
	{  
		PERSISION _x = (id % w - (PERSISION)w / 2.0) * zoom + x;
		PERSISION _y = -(id / w - (PERSISION)h / 2.0) * zoom + y;
		PERSISION tempX = 0;
		int iterations = ITERATIONS;

		x = var_x;
		y = var_y;

		// if (_x == _y) { iterations = 0; }

		while (iterations > 0)
		{
			// if ((id % w - (double)w / 2.0) + 30 < (double)w) { iterations = 0; break; }

			/*tempX = (x * x * x - x * y * y) / (1 + x * x - y) + _x;
			y = (3 * x * x * y - y * y) / (1 + x * x - y) + _y;
			x = tempX;*/

			if (mandel)
			{
				tempX = _x + x * x - y * y;
				y = _y + 2 * x * y;
				x = tempX;

				if (x * x + y * y > 4.0) { break; }
			}
			else
			{
				tempX = _x * _x - _y * _y - var_x;
				_y = 2.0f * _x * _y - var_y;
				_x = tempX;

				if (_x * _x + _y * _y > 4.0) { break; }
			}

			/*tempX = _x + x * x - y * y;
			y = _y + 2 * abs(x * y);
			x = tempX;*/

			/*tempX = _x * _x - _y * _y - var_x;
			_y = 2.0f * _x * _y - var_y; 
			_x = tempX;*/

			
			iterations--;
		}

		d_outPut[id] = (int)(iterations / (float)ITERATIONS * 255);
	}
}


Fractal::Fractal(int w, int h)
{
	this->w = w; this->h = h;
	blockSize = BLOCK_SIZE;
	blockAmount = (int)ceil((float)w * (float)h / (float)BLOCK_SIZE);

	taskRunning = false;
	d_outPut = nullptr;

	h_outPut = new int[w * h];
	hipMalloc((void**)&d_outPut, sizeof(int) * w * h);
}


Fractal::~Fractal()
{
	FreeMemory();
}


void Fractal::changeArea(int _w, int _h)
{
	FreeMemory();
	w = _w; h = _h;
	h_outPut = new int[w * h];
	hipMalloc((void**)&d_outPut, sizeof(int) * w * h);
	hipDeviceSynchronize();

	blockAmount = (int)ceil((float)w * (float)h / (float)BLOCK_SIZE);
}


void Fractal::FreeMemory()
{
	if (d_outPut != nullptr && h_outPut != nullptr)
	{
		hipDeviceSynchronize();
		hipFree(d_outPut);
		delete[] h_outPut;
		d_outPut = nullptr;
		h_outPut = nullptr;
		hipDeviceSynchronize();
	}
}


void Fractal::calculate(PERSISION x, PERSISION y, PERSISION zoom, PERSISION var_x, PERSISION var_y, bool mandel)
{
	kernel <<< blockAmount, blockSize >>> (w, h, x, y, zoom, var_x, var_y, mandel, d_outPut);
	hipDeviceSynchronize();

	hipMemcpy(h_outPut, d_outPut, sizeof(int) * w * h, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}